#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/inner_product_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void InnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weight, bottom_data, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            this->blobs_[1]->gpu_data(), top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans,
                          transpose_ ? CblasNoTrans : CblasTrans,
                          M_, N_, K_, (Dtype)1.,
                          bottom_data, weight, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
  }

}

//for iccv17 bier loss
template <typename Dtype>
__global__ void compute_norm(const int d, const int h, Dtype* w, Dtype* temp) {
  CUDA_KERNEL_LOOP(index, d) {
        for(int i = 0; i < h; i++)
        {
               temp[index]=temp[index] + w[index*h+i] * w[index*h+i];
        }
        temp[index] = temp[index] - Dtype(1.0);
  }
}
template <typename Dtype>
__global__ void compute_gradients(const int d, const int h, Dtype* w, Dtype* w_g, Dtype* temp, Dtype lamda) {
  CUDA_KERNEL_LOOP(index, d*h) {
        int i = index/h;
        int j = index%h;
        w_g[i*h+j] = w_g[i*h+j] + Dtype(4.0) * lamda * temp[i] * w[i*h+j];
  }
}


template <typename Dtype>
void InnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if(this->layer_param_.inner_product_param().param_propagate_down()){
  
  if (this->param_propagate_down_[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    // Gradient with respect to weight
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          K_, N_, M_,
          (Dtype)1., bottom_data, top_diff,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          N_, K_, M_,
          (Dtype)1., top_diff, bottom_data,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    }
    //for iccv17 BIER LOSS init
    if(bier_init_)
    {
        caffe_gpu_set(bier_temp_.count(), Dtype(0.0), bier_temp_.mutable_gpu_data());
        compute_norm<Dtype><<<CAFFE_GET_BLOCKS(N_), CAFFE_CUDA_NUM_THREADS>>>(N_, K_, this->blobs_[0]->mutable_gpu_data(), bier_temp_.mutable_gpu_data());
        //compute_gradients<Dtype><<<CAFFE_GET_BLOCKS(N_*K_), CAFFE_CUDA_NUM_THREADS>>>(N_, K_, this->blobs_[0]->mutable_gpu_data(), this->blobs_[0]->mutable_gpu_diff(), bier_temp_.mutable_gpu_data(), lamda_);
        //add new codes
        compute_gradients<Dtype><<<CAFFE_GET_BLOCKS(N_*K_), CAFFE_CUDA_NUM_THREADS>>>(N_, K_, this->blobs_[0]->mutable_gpu_data(), this->blobs_[0]->mutable_gpu_diff(), bier_temp_.mutable_gpu_data(), lamda_/Dtype(N_));
    }
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
  }
  
  }
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bottom data
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
          M_, K_, N_,
          (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
          (Dtype)0., bottom[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
          M_, K_, N_,
         (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
         (Dtype)0., bottom[0]->mutable_gpu_diff());
    }
  }

}

INSTANTIATE_LAYER_GPU_FUNCS(InnerProductLayer);

}  // namespace caffe
