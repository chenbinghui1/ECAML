#include "hip/hip_runtime.h"
// add by Binbin Xu
// declanxu@gmail.com or declanxu@126.com
// Zhejiang University, State Key Lab of CAD&CG.


#include <algorithm>
#include <cfloat>
#include <vector>

// #include "thrust/device_vector.h"
#include "caffe/util/io.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/normalization_layer.hpp"

namespace caffe {

template <typename Dtype>
void NormalizationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype normsqr;
  int n = bottom[0]->num();
  int d = bottom[0]->count() / n;
  caffe_gpu_powx(n*d, bottom_data, Dtype(2), squared_data);
  for (int i=0; i<n; ++i) {
    caffe_gpu_asum<Dtype>(d, squared_data+i*d, &normsqr);
    caffe_gpu_scale<Dtype>(d, pow(normsqr, -0.5), bottom_data+i*d, top_data+i*d);
  }
/*
  const Dtype* out = top[0]->cpu_data();
  for (int i=0; i<n; ++i) {
    int ptr = i*d;
    //Dtype tmp = 0.0;
    std::cout << i << ": ";
    for (int j=0; j < d; ++j) {
	//tmp += out[ptr]*out[ptr++];
	std::cout << out[ptr++] << " ";
    }
    std::cout << "\n";
//    LOG(INFO) << i << ": " << tmp;
  }
*/
}
  
template <typename Dtype>
void NormalizationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int n = top[0]->num();
  int d = top[0]->count() / n;
  Dtype a;
  for (int i=0; i<n; ++i) {
    caffe_gpu_dot(d, top_data+i*d, top_diff+i*d, &a);
    caffe_gpu_scale(d, a, top_data+i*d, bottom_diff+i*d);
    caffe_gpu_sub(d, top_diff+i*d, bottom_diff+i*d, bottom_diff+i*d);
    caffe_gpu_dot(d, bottom_data+i*d, bottom_data+i*d, &a);
    caffe_gpu_scale(d, Dtype(pow(a, -0.5)), bottom_diff+i*d, bottom_diff+i*d);
  }
/*
const Dtype* b = bottom[0]->cpu_data();
for (int i = 0; i < n; i++) {
    std::cout << i << ": ";
    int tmp = i*128;
    for (int j = 0; j < 128; j++) {
	std::cout << b[tmp++] << " ";
    }
    std::cout << "\n:";
}
*/
}

// INSTANTIATE_CLASS(NormalizationLayer);

INSTANTIATE_LAYER_GPU_FUNCS(NormalizationLayer);
}  // namespace caffe
